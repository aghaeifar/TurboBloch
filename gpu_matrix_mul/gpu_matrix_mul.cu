// gpu_matrix_mul.cpp : Defines the functions for the static library.
//

#include "pch.h"
#include "framework.h"
#include "gpu_matrix_mul.h"

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <iostream>

//  mexcuda -L'C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v11.4\lib\x64' -lcublas -lcudart ...

// TODO: This is an example of a library function
int gpu_matrix_mul::mul_matmat(std::complex<double> *mat_in1, std::complex<double> *mat_in2, std::complex<double> *mat_out, size_t row, size_t col_row, size_t col)
{
	//Allocate space for device copies in device memory
	hipDoubleComplex* cdc_mat_in1;
	hipDoubleComplex* cdc_mat_in2;
	hipDoubleComplex* cdc_mat_out;
	
	hipMalloc(&cdc_mat_in1, row * col_row * sizeof(hipDoubleComplex));
	hipMalloc(&cdc_mat_in2, col_row * col * sizeof(hipDoubleComplex));
	hipMalloc(&cdc_mat_out, row * col * sizeof(hipDoubleComplex));

	if (hipMemcpy(cdc_mat_in1, mat_in1, row * col_row * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) != hipSuccess)
		std::cout << "copying from host to device failed" << std::endl;
	if (hipMemcpy(cdc_mat_in2, mat_in2, col_row * col * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) != hipSuccess)
		std::cout << "copying from host to device failed" << std::endl;
	
	
	// https://docs.nvidia.com/cuda/cublas/index.html
	// https://stackoverflow.com/questions/43441573/using-cublas-with-complex-numbers-from-thrust
	hipblasHandle_t handle;
	hipblasStatus_t stat = hipblasCreate(&handle);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		std::cout<<"CUBLAS initialization failed"<<std::endl;
		return EXIT_FAILURE;
	}
	std::complex<double> alpha(1.0, 0.0);
	std::complex<double> beta(0.0, 0.0);

	hipDoubleComplex* _alpha = reinterpret_cast<hipDoubleComplex*>(&alpha);
	hipDoubleComplex* _beta = reinterpret_cast<hipDoubleComplex*>(&beta);

	if(hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, row, col, col_row,
				_alpha, cdc_mat_in1, row, cdc_mat_in2, col_row, _beta, cdc_mat_out, row) != HIPBLAS_STATUS_SUCCESS)
		std::cout << "CUBLAS failed to multiply" << std::endl;

	if (hipMemcpy(mat_out, cdc_mat_out, row * col * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost) != hipSuccess)
		std::cout << "copying from device to host failed" << std::endl;

	hipblasDestroy(handle);
	hipFree(cdc_mat_in1);
	hipFree(cdc_mat_in2);
	hipFree(cdc_mat_out);
    
	cdc_mat_in1 = cdc_mat_in2 = cdc_mat_out = NULL;
    return EXIT_SUCCESS;
}

int gpu_matrix_mul::mul_matmat(std::complex<float> *mat_in1, std::complex<float> *mat_in2, std::complex<float> *mat_out, size_t row, size_t col_row, size_t col)
{
    hipFloatComplex* cdc_mat_in1;
    hipFloatComplex* cdc_mat_in2;
    hipFloatComplex* cdc_mat_out;

    hipMalloc(&cdc_mat_in1, row * col_row * sizeof(hipFloatComplex));
    hipMalloc(&cdc_mat_in2, col_row * col * sizeof(hipFloatComplex));
    hipMalloc(&cdc_mat_out, row * col * sizeof(hipFloatComplex));

    if (hipMemcpy(cdc_mat_in1, mat_in1, row * col_row * sizeof(hipFloatComplex), hipMemcpyHostToDevice) != hipSuccess)
        std::cout << "copying from host to device failed" << std::endl;
    if (hipMemcpy(cdc_mat_in2, mat_in2, col_row * col * sizeof(hipFloatComplex), hipMemcpyHostToDevice) != hipSuccess)
        std::cout << "copying from host to device failed" << std::endl;


    // https://docs.nvidia.com/cuda/cublas/index.html
    // https://stackoverflow.com/questions/43441573/using-cublas-with-complex-numbers-from-thrust
    hipblasHandle_t handle;
    hipblasStatus_t stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        std::cout<<"CUBLAS initialization failed"<<std::endl;
        return EXIT_FAILURE;
    }
    std::complex<float> alpha(1.0, 0.0);
    std::complex<float> beta(0.0, 0.0);

    hipFloatComplex* _alpha = reinterpret_cast<hipFloatComplex*>(&alpha);
    hipFloatComplex* _beta = reinterpret_cast<hipFloatComplex*>(&beta);

    if(hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, row, col, col_row,
                _alpha, cdc_mat_in1, row, cdc_mat_in2, col_row, _beta, cdc_mat_out, row) != HIPBLAS_STATUS_SUCCESS)
        std::cout << "CUBLAS failed to multiply" << std::endl;

    if (hipMemcpy(mat_out, cdc_mat_out, row * col * sizeof(hipFloatComplex), hipMemcpyDeviceToHost) != hipSuccess)
        std::cout << "copying from device to host failed" << std::endl;

    hipblasDestroy(handle);
    hipFree(cdc_mat_in1);
    hipFree(cdc_mat_in2);
    hipFree(cdc_mat_out);

    cdc_mat_in1 = cdc_mat_in2 = cdc_mat_out = NULL;
    return EXIT_SUCCESS;
}

gpu_matrix_mul::gpu_matrix_mul()
{

}

gpu_matrix_mul::~gpu_matrix_mul()
{

}
